#include "hip/hip_runtime.h"
#include "estimator.h"

// using namespace Eigen;
using namespace std;

#define CUDA_CHECK(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while (0)


// CUDA Kernel for element-wise operations
__global__ void compute_products(const float* estimate, const float* target, const float* valid, 
                                  float* valid_estimate_estimate_sum, float* valid_estimate_sum, float* valid_sum,
                                  float* valid_estimate_target_sum, float* valid_target_sum, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        float val = valid[idx];
        float est = estimate[idx];
        float tgt = target[idx];

        atomicAdd(valid_estimate_estimate_sum, val * est * est);
        atomicAdd(valid_estimate_sum, val * est);
        atomicAdd(valid_sum, val);
        atomicAdd(valid_estimate_target_sum, val * est * tgt);
        atomicAdd(valid_target_sum, val * tgt);
    }
}

__global__ void clampKernel(float* data, int rows, int cols, size_t pitch, float minVal, float maxVal) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        float* row = (float*)((char*)data + y * pitch);
        row[x] = fminf(fmaxf(row[x], minVal), maxVal);
    }
}

namespace depthpipe {

// // Function to compute scale and shift using least squares
// pair<ArrayXf, ArrayXf> compute_scale_and_shift_ls(const ArrayXXf& prediction, 
//                                                   const ArrayXXf& target, 
//                                                   const ArrayXXf& mask) {
//     // Sum axes (equivalent to numpy sum over axis 0, 1)
//     ArrayXf a_00 = (mask * prediction * prediction).colwise().sum();
//     ArrayXf a_01 = (mask * prediction).colwise().sum();
//     ArrayXf a_11 = mask.colwise().sum();

//     ArrayXf b_0 = (mask * prediction * target).colwise().sum();
//     ArrayXf b_1 = (mask * target).colwise().sum();

//     ArrayXf x_0 = ArrayXf::Zero(b_0.size());
//     ArrayXf x_1 = ArrayXf::Zero(b_1.size());

//     // Calculate determinant
//     ArrayXf det = a_00 * a_11 - a_01 * a_01;
//     Array<bool, Dynamic, 1> valid = det > 0;

//     // Scale and shift computation
//     x_0 = (a_11 * b_0 - a_01 * b_1).array() / det;
//     x_1 = (-a_01 * b_0 + a_00 * b_1).array() / det;

//     // Apply validity mask
//     x_0 = valid.select(x_0, 0);
//     x_1 = valid.select(x_1, 0);

//     return {x_0, x_1};
// }

// LeastSquaresEstimator::LeastSquaresEstimator(const ArrayXXf& estimate, const ArrayXXf& target, const ArrayXXf& valid)
    // : estimate(estimate), target(target), valid(valid), scale(1.0), shift(0.0) {}
LeastSquaresEstimator::LeastSquaresEstimator(const float* estimate, const float* target, const float* valid, int rows, int cols)
    : estimate(estimate), target(target), valid(valid), rows(rows), cols(cols), scale(1.0), shift(0.0) {}


// Method to compute scale and shift
// TODO: Currently only supports batch size of 1
// void LeastSquaresEstimator::compute_scale_and_shift() {
//     // tie(scale, shift) = compute_scale_and_shift_ls(estimate, target, valid);

//     // Sum axes (equivalent to numpy sum over axis 0, 1)
//     // ArrayXf a_00 = (valid * estimate * estimate).colwise().sum();
//     // ArrayXf a_01 = (valid * estimate).colwise().sum();
//     // ArrayXf a_11 = valid.colwise().sum();
//     float a_00 = (valid * estimate * estimate).sum();
//     float a_01 = (valid * estimate).sum();
//     float a_11 = valid.sum();

//     float b_0 = (valid * estimate * target).sum();
//     float b_1 = (valid * target).sum();
//     // cout << "b_0: " << b_0 << endl;
//     // cout << "b_1: " << b_1 << endl;

//     float x_0 = 0.0;
//     float x_1 = 0.0;

//     // Calculate determinant
//     float det = a_00 * a_11 - a_01 * a_01;
//     // Array<bool, Dynamic, 1> mask = det > 0;
//     if (det <= 0) {
//         cout << "Matrix not positive definite. Not scaling depth." << endl;
//         scale = 1.0;
//         shift = 0.0;
//         return;
//     }

//     // Scale and shift computation
//     x_0 = (a_11 * b_0 - a_01 * b_1) / det;
//     x_1 = (-a_01 * b_0 + a_00 * b_1) / det;

//     scale = x_0;
//     shift = x_1;
// }

// Compute scale and shift using CUDA
void LeastSquaresEstimator::compute_scale_and_shift() {
    // Allocate GPU memory
    float *d_estimate, *d_target, *d_valid;
    float *d_valid_estimate_estimate_sum, *d_valid_estimate_sum, *d_valid_sum, *d_valid_target_sum, *d_valid_estimate_target_sum;
    CUDA_CHECK(hipMalloc(&d_estimate, rows * cols * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_target, rows * cols * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_valid, rows * cols * sizeof(float)));

    CUDA_CHECK(hipMalloc(&d_valid_estimate_estimate_sum,  sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_valid_estimate_sum, sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_valid_sum, sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_valid_target_sum, sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_valid_estimate_target_sum, sizeof(float)));

    // Copy input data to GPU
    CUDA_CHECK(hipMemcpy(d_estimate, estimate, rows * cols * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_target, target, rows * cols * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_valid, valid, rows * cols * sizeof(float), hipMemcpyHostToDevice));

    // Initialize output arrays on GPU
    CUDA_CHECK(hipMemset(d_valid_estimate_estimate_sum, 0, sizeof(float)));
    CUDA_CHECK(hipMemset(d_valid_estimate_sum, 0, sizeof(float)));
    CUDA_CHECK(hipMemset(d_valid_sum, 0, sizeof(float)));
    CUDA_CHECK(hipMemset(d_valid_target_sum, 0, sizeof(float)));
    CUDA_CHECK(hipMemset(d_valid_estimate_target_sum, 0, sizeof(float)));

    // Launch kernel
    int threads = 256;
    int blocks = (rows * cols + threads - 1) / threads;
    compute_products<<<blocks, threads>>>(d_estimate, d_target, d_valid, 
                                          d_valid_estimate_estimate_sum, d_valid_estimate_sum, d_valid_sum,
                                          d_valid_estimate_target_sum, d_valid_target_sum, rows, cols);
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy results back to CPU
    float a_00, a_01, a_11, b_0, b_1;
    CUDA_CHECK(hipMemcpy(&a_00, d_valid_estimate_estimate_sum, sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&a_01, d_valid_estimate_sum, sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&a_11, d_valid_sum, sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&b_0, d_valid_estimate_target_sum, sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&b_1, d_valid_target_sum, sizeof(float), hipMemcpyDeviceToHost));

//     float a_00 = (valid * estimate * estimate).sum();
//     float a_01 = (valid * estimate).sum();
//     float a_11 = valid.sum();

//     float b_0 = (valid * estimate * target).sum();
//     float b_1 = (valid * target).sum();

    // Perform final reduction on CPU
    // float a_00 = 0, a_01 = 0, a_11 = 0, b_0 = 0, b_1 = 0;

    // Compute determinant and scale/shift
    // float det = a_00 * a_11 - a_01 * a_01;
    // if (det <= 0) {
    //     std::cerr << "Matrix not positive definite. Not scaling depth." << std::endl;
    //     scale = 1.0;
    //     shift = 0.0;
    // } else {
    //     scale = (a_11 * b_0 - a_01 * b_1) / det;
    //     shift = (-a_01 * b_0 + a_00 * b_1) / det;
    // }

    float x_0 = 0.0;
    float x_1 = 0.0;

    // Calculate determinant
    float det = a_00 * a_11 - a_01 * a_01;
    // Array<bool, Dynamic, 1> mask = det > 0;
    if (det <= 0) {
        cout << "Matrix not positive definite. Not scaling depth." << endl;
        scale = 1.0;
        shift = 0.0;
        return;
    }

    // Scale and shift computation
    x_0 = (a_11 * b_0 - a_01 * b_1) / det;
    x_1 = (-a_01 * b_0 + a_00 * b_1) / det;

    scale = x_0;
    shift = x_1;

    // Free GPU memory
    hipFree(d_estimate);
    hipFree(d_target);
    hipFree(d_valid);
    hipFree(d_valid_estimate_estimate_sum);
    hipFree(d_valid_estimate_sum);
    hipFree(d_valid_sum);
    hipFree(d_valid_target_sum);
    hipFree(d_valid_estimate_target_sum);
}

// Apply scale and shift to the estimate
// void LeastSquaresEstimator::apply_scale_and_shift() {
    // output = estimate * scale + shift;
    // output.create(relDepthMap.size(), relDepthMap.type());
// }


// Clamp output values between specified minimum and maximum values
void LeastSquaresEstimator::clamp_min_max(cv::cuda::GpuMat &input, float clamp_min, float clamp_max, bool inverted)
{
    // Launch kernel
    dim3 threads(16, 16);
    dim3 blocks((cols + threads.x - 1) / threads.x, (rows + threads.y - 1) / threads.y);

    if (inverted)
        clampKernel<<<blocks, threads>>>(input.ptr<float>(), rows, cols, input.step, 1.0f/clamp_max, 1.0f/clamp_min);
    else
        clampKernel<<<blocks, threads>>>(input.ptr<float>(), rows, cols, input.step, clamp_min, clamp_max);
}

// Clamp output values between specified minimum and maximum values
// void LeastSquaresEstimator::clamp_min_max(float clamp_min, float clamp_max, bool inverted) {
    // if (output.size() == 0) {
        // cerr << "Output not initialized, call apply_scale_and_shift() first!" << endl;
        // return;
    // }

//     if (clamp_min > 0.0f) {
//         if (inverted) {
//             float clamp_min_inv = 1.0f / clamp_min;
//             output = output.min(clamp_min_inv);
//         }
//         else
//             output = output.max(clamp_min);
//     }
    
//     if (clamp_max > 0.0f) {
//         if (inverted) {
//             float clamp_max_inv = 1.0f / clamp_max;
//             output = output.max(clamp_max_inv);
//         }
//         else
//             output = output.min(clamp_max);
//     }
// }

} // namespace depthpipe